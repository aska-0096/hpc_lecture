
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__device__ __managed__ int bucket[5];

__global__ void initialization(int * bucket){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  bucket[i] = 0;
}
__global__ void reduction(int *bucket,int *key){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  atomicAdd(&bucket[key[i]],1);
}

__global__ void  bucketsort(int *bucket, int *key){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for(int j=0, k=0;k<=i;j++){
  key[i]=j;
  k+=bucket[j];  
}
} 


int main() {
  int n = 50;
  int range = 5;
  int *key;
  hipMallocManaged(&key, n*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");
  initialization<<<1,range>>>(bucket);
  hipDeviceSynchronize();
  reduction<<<1,n>>>(bucket, key);
  hipDeviceSynchronize();
  bucketsort<<<1,n>>>(bucket, key);
  hipDeviceSynchronize();

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
}
